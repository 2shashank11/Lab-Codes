#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void odd_even_phase(int *arr, int n, int phase) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < n - 1) {
        if ((idx % 2 == phase) && (idx < n - 1)) {
            if (arr[idx] > arr[idx + 1]) {
                int temp = arr[idx];
                arr[idx] = arr[idx + 1];
                arr[idx + 1] = temp;
            }
        }
    }
}

void odd_even_transposition_sort(int *arr, int n) {
    for (int phase = 0; phase < n; phase++) {
        odd_even_phase<<<(n + 255) / 256, 256>>>(arr, n, phase % 2);

        hipDeviceSynchronize();
    }
}

int main() {
    int n;

    printf("Enter the number of elements: ");
    scanf("%d", &n);

    int *h_arr = (int *)malloc(n * sizeof(int));

    printf("Enter the elements: ");
    for (int i = 0; i < n; i++) {
        scanf("%d", &h_arr[i]);
    }

    int *d_arr;
    hipMalloc((void**)&d_arr, n * sizeof(int));

    hipMemcpy(d_arr, h_arr, n * sizeof(int), hipMemcpyHostToDevice);

    odd_even_transposition_sort(d_arr, n);

    hipMemcpy(h_arr, d_arr, n * sizeof(int), hipMemcpyDeviceToHost);

    printf("Sorted Array: ");
    for (int i = 0; i < n; i++) {
        printf("%d ", h_arr[i]);
    }
    printf("\n");

    hipFree(d_arr);
    free(h_arr);

    return 0;
}


// Enter the number of elements: 5
// Enter the elements: 6 2 13 7 5
// Sorted Array: 2 5 6 7 13 