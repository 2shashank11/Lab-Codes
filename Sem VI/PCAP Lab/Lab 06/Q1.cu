#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>


__global__ void convolution_1D_basic_kernel(float *N, float *M, float *P, int Mask_Width, int Width){
    int i = blockIdx.x*blockDim.x+threadIdx.x;

    float Pvalue = 0;
    int N_start_point = i-(Mask_Width/2);
    for(int j=0; j<Mask_Width; j++){
        if(N_start_point + j >= 0 && N_start_point + j < Width){
            Pvalue += N[N_start_point + j] * M[j];
        }
    }
    P[i] = Pvalue;
}

int main(void){
    int n, w;
    float *a, *m, *p;
    float *d_a, *d_p, *d_m;

    printf("Enter size of array: ");
    scanf("%d", &n);
    a = (float*)(malloc(n*sizeof(float)));
    p = (float*)(malloc(n*sizeof(float)));
    hipMalloc((void**)&d_a, n*sizeof(float));
    hipMalloc((void**)&d_p, n*sizeof(float));

    printf("Enter %d elements of the array: ", n);
    for(int i=0; i<n; i++){
        scanf("%f", &a[i]);
    }

    printf("Enter width of the mask: ");
    scanf("%d", &w);
    m = (float*)(malloc(w*sizeof(float)));
    hipMalloc((void**)&d_m, w*sizeof(float));

    printf("Enter %d elements of the mask: ", w);
    for(int i=0; i<w; i++){
        scanf("%f", &m[i]);
    }

    hipMemcpy(d_a, a, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_m, m, w*sizeof(float), hipMemcpyHostToDevice);

    convolution_1D_basic_kernel<<<1, n>>>(d_a, d_m, d_p, w, n);
    hipMemcpy(p, d_p, n*sizeof(float), hipMemcpyDeviceToHost);

    printf("Resultant array is: ");
    for(int i=0; i<n; i++){
        printf("%.2f ", p[i]);
    }
    printf("\n");

    return 0;
}


// Enter size of array: 7
// Enter 7 elements of the array: 1 2 3 4 5 6 7
// Enter width of the mask: 5
// Enter 5 elements of the mask: 3 4 5 4 3
// Resultant array is: 22.00 38.00 57.00 76.00 95.00 90.00 74.00 