#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void parallel_selection_sort(int *arr, int n, int step) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx >= step && idx < n) {
        int min_idx = step;

        for (int i = step + 1; i < n; i++) {
            if (arr[i] < arr[min_idx]) {
                min_idx = i;
            }
        }

        if (min_idx != step) {
            int temp = arr[step];
            arr[step] = arr[min_idx];
            arr[min_idx] = temp;
        }
    }
}

void selection_sort_parallel(int *arr, int n) {
    for (int step = 0; step < n - 1; step++) {
        parallel_selection_sort<<<(n + 255) / 256, 256>>>(arr, n, step);

        hipDeviceSynchronize();
    }
}

int main() {
    int n;

    printf("Enter the number of elements: ");
    scanf("%d", &n);

    int *h_arr = (int *)malloc(n * sizeof(int));

    printf("Enter the elements: ");
    for (int i = 0; i < n; i++) {
        scanf("%d", &h_arr[i]);
    }

    int *d_arr;
    hipMalloc((void**)&d_arr, n * sizeof(int));

    hipMemcpy(d_arr, h_arr, n * sizeof(int), hipMemcpyHostToDevice);

    selection_sort_parallel(d_arr, n);

    hipMemcpy(h_arr, d_arr, n * sizeof(int), hipMemcpyDeviceToHost);

    printf("Sorted Array: ");
    for (int i = 0; i < n; i++) {
        printf("%d ", h_arr[i]);
    }
    printf("\n");

    hipFree(d_arr);
    free(h_arr);

    return 0;
}


// Enter the number of elements: 5
// Enter the elements: 6 2 13 7 5
// Sorted Array: 2 5 6 7 13 