#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

__device__ bool wordMatch(const char* sentence, int index, const char* word) {
    int i = 0;
    while (word[i] != '\0' && sentence[index + i] != '\0') {
        if (sentence[index + i] != word[i]) {
            return false;
        }
        i++;
    }
    return word[i] == '\0' && (sentence[index + i] == ' ' || sentence[index + i] == '\0');
}

__global__ void countWordOccurrences(const char* sentence, const char* word, int sentenceLength, int wordLength, int* count) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < sentenceLength - wordLength + 1) {
        if (wordMatch(sentence, idx, word)) {
            atomicAdd(count, 1);
        }
    }
}

int main() {

    char sentence[1024];
    char word[100];

    printf("Enter sentence: ");
    fgets(sentence, sizeof(sentence), stdin);
    sentence[strcspn(sentence, "\n")] = 0; 
    printf("Enter word: ");
    fgets(word, sizeof(word), stdin);
    word[strcspn(word, "\n")] = 0;

    int sentenceLength = strlen(sentence);
    int wordLength = strlen(word);

    char* d_sentence;
    char* d_word;
    int* d_count;
    int count = 0;

    hipMalloc(&d_sentence, sentenceLength + 1);
    hipMalloc(&d_word, wordLength + 1);
    hipMalloc(&d_count, sizeof(int));

    hipMemcpy(d_sentence, sentence, sentenceLength + 1, hipMemcpyHostToDevice);
    hipMemcpy(d_word, word, wordLength + 1, hipMemcpyHostToDevice);
    hipMemcpy(d_count, &count, sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (sentenceLength + blockSize - 1) / blockSize;

    countWordOccurrences<<<gridSize, blockSize>>>(d_sentence, d_word, sentenceLength, wordLength, d_count);

    hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);

    printf("The word %s appears %d times in the sentence\n", word, count);

    hipFree(d_sentence);
    hipFree(d_word);
    hipFree(d_count);

    return 0;
}


// Enter sentence: hello world
// Enter word: hello
// The word hello appears 1 times in the sentence