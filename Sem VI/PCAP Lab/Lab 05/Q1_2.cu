#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void vectadd(int *a , int*b , int* c , int n){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid<n){
	c[tid] = a[tid] + b[tid];
	}
}

int main(void){
	int *a,*b,*c ,*d , *e;
	int *d_a , *d_b , *d_c , *d_d , *d_e;
	int n;
	printf("Enter length of vector: ");
	scanf("%d",&n);
	int size = n* sizeof(int);

	a = (int*)malloc(size);
    b = (int*)malloc(size);
    c = (int*)malloc(size);
    d = (int*)malloc(size);
    e = (int*)malloc(size);

	hipMalloc((void**)&d_a , size);
	hipMalloc((void**)&d_b , size);
	hipMalloc((void**)&d_c , size);
	hipMalloc((void**)&d_d , size);
	hipMalloc((void**)&d_e , size);

	printf("Enter first vector : ");
	for(int i=0;i<n;i++){
	scanf("%d",&a[i]);
	}
	printf("Enter second vector : ");
	for(int i=0;i<n;i++){
	scanf("%d",&b[i]);
	}

	hipMemcpy(d_a , a ,size, hipMemcpyHostToDevice);
	hipMemcpy(d_b , b ,size, hipMemcpyHostToDevice);

	//n threads (n threads in one block)
	printf("n threads (n threads in one block)\n");
	vectadd<<<1 , n>>>(d_a , d_b , d_c , n);
	hipMemcpy(c , d_c , size ,hipMemcpyDeviceToHost);
	printf("Result: ");
	for(int i=0;i<n;i++){
	printf("%d ",c[i]);
	}
	printf("\n");

	//n blocks(1 thread per block )
	printf("n blocks(1 thread per block )\n");
	vectadd<<<n , 1>>>(d_a , d_b , d_d , n);
	hipMemcpy(d , d_d , size ,hipMemcpyDeviceToHost);
	printf("Result: ");
	for(int i=0;i<n;i++){
	printf("%d ",d[i]);
	}

	printf("\n");

	dim3 dimGrid(ceil(n/256.0) , 1 ,1);
	dim3 dimBlock(256,1,1);
	//256 threads per block , varying no of blocks 
	printf("256 threads per block , varying no of blocks\n");
	vectadd<<<dimGrid , dimBlock>>>(d_a , d_b , d_e , n);
	hipMemcpy(e , d_e , size ,hipMemcpyDeviceToHost);
	printf("Result: ");
	for(int i=0;i<n;i++){
	printf("%d ",e[i]);
	}
	printf("\n");

    free(a);
    free(b);
    free(c);
    free(d);
    free(e);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	hipFree(d_d);
	hipFree(d_e);
	return 0;
}

// Enter length of vector: 5
// Enter first vector : 1 2 3 4 5
// Enter second vector : 5 6 7 8 9
// n threads (n threads in one block)
// Result: 6 8 10 12 14 
// n blocks(1 thread per block )
// Result: 6 8 10 12 14 
// 256 threads per block , varying no of blocks
// Result: 6 8 10 12 14 