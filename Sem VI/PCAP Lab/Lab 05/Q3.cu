#include "hip/hip_runtime.h"

#include <stdio.h>
#include<math.h>

__global__ void findsin(double *a , double* c , int n){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid<n){
    c[tid] = sin(a[tid]);
    }
}

int main(void){
    double *a,*c;
    double *d_a , *d_c;
    int n;
    printf("Enter length of vector: ");
    scanf("%d",&n);
    double size = n* sizeof(double);

    a = (double*)malloc(size);
    c = (double*)malloc(size);

    hipMalloc((void**)&d_a , size);
    hipMalloc((void**)&d_c , size);

    printf("Enter vector containing angles in radians: ");
    for(int i=0;i<n;i++){
    scanf("%lf",&a[i]);
    }

    hipMemcpy(d_a , a ,size, hipMemcpyHostToDevice);

    dim3 dimGrid(ceil(n/256.0) , 1 ,1);
    dim3 dimBlock(256,1,1);
    //256 threads per block , varying no of blocks 
    printf("256 threads per block , varying no of blocks\n");
    findsin<<<dimGrid , dimBlock>>>(d_a ,d_c , n);
    hipMemcpy(c , d_c , size ,hipMemcpyDeviceToHost);
    printf("Result: ");
    for(int i=0;i<n;i++){
    printf("%.4f\t",c[i]);
    }
    printf("\n");

    free(a);
    free(c);
    
    hipFree(d_a);
    hipFree(d_c);
    return 0;
}

// Enter length of vector: 5
// Enter vector containing angles in radians: 1 2 3 4 5
// 256 threads per block , varying no of blocks
// Result: 0.8415  0.9093  0.1411  -0.7568 -0.9589