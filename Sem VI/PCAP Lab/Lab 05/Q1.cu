
#include <hip/hip_runtime.h>
#include <stdio.h>

const short N = 5 ;

__global__ void Vector_Addition_Block ( const int *dev_a , const int *dev_b , int *dev_c_block){
    unsigned short tid = blockIdx.x ;
     
    if ( tid < N ){
        dev_c_block [tid] = dev_a[tid] + dev_b[tid] ;
    }
}

__global__ void Vector_Addition_Thread ( const int *dev_a , const int *dev_b , int *dev_c_thread){
    unsigned short tid = threadIdx.x ;
     
    if ( tid < N ){
        dev_c_thread [tid] = dev_a[tid] + dev_b[tid] ;
    }

}


int main (void){

    int Host_a[N], Host_b[N], Host_c_thread[N], Host_c_block[N];

    int *dev_a , *dev_b, *dev_c_thread, *dev_c_block ;

    hipMalloc((void **)&dev_a , N*sizeof(int) );
    hipMalloc((void **)&dev_b , N*sizeof(int) );
    hipMalloc((void **)&dev_c_thread , N*sizeof(int) );
    hipMalloc((void **)&dev_c_block , N*sizeof(int) );

    for ( int i = 0; i <N ; i++ ){
        Host_a[i] = -i ;
        Host_b[i] = i*i ; 
    }

    hipMemcpy (dev_a , Host_a , N*sizeof(int) , hipMemcpyHostToDevice);
    hipMemcpy (dev_b , Host_b , N*sizeof(int) , hipMemcpyHostToDevice);

    //BLOCK
    printf("\nBlock Vector Sum: \n");
    Vector_Addition_Block <<< N, 1  >>> (dev_a , dev_b , dev_c_block ) ;

    hipMemcpy(Host_c_block , dev_c_block , N*sizeof(int) , hipMemcpyDeviceToHost);

    for ( int i = 0; i<N; i++ ){
        printf ("%d + %d = %d\n", Host_a[i] , Host_b[i] , Host_c_block[i] ) ;
    }

    //THREAD
    printf("\nThread Vector Sum: \n");
    Vector_Addition_Thread <<< 1, N  >>> (dev_a , dev_b , dev_c_thread ) ;

    hipMemcpy(Host_c_thread , dev_c_thread , N*sizeof(int) , hipMemcpyDeviceToHost);

    for ( int i = 0; i<N; i++ ){
        printf ("%d + %d = %d\n", Host_a[i] , Host_b[i] , Host_c_thread[i] ) ;
    }


    hipFree (dev_a) ;
    hipFree (dev_b) ;
    hipFree (dev_c_block) ;
    hipFree(dev_c_thread);

    return 0 ;
}