#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 4

__global__ void matrixAddRow(int *A, int *B, int *C, int width) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < width) {
        for (int col = 0; col < width; col++) {
            C[row * width + col] = A[row * width + col] + B[row * width + col];
        }
    }
}

__global__ void matrixAddCol(int *A, int *B, int *C, int width) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < width) {
        for (int row = 0; row < width; row++) {
            C[row * width + col] = A[row * width + col] + B[row * width + col];
        }
    }
}

__global__ void matrixAddElement(int *A, int *B, int *C, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < width && col < width) {
        C[row * width + col] = A[row * width + col] + B[row * width + col];
    }
}

int main() {
    int A[N][N], B[N][N], C[N][N];
    int *d_A, *d_B, *d_C;

    printf("Enter matrix 1: \n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            scanf("%d", &A[i][j]);
        }
    }

    printf("Enter matrix 2: \n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            scanf("%d", &B[i][j]);
        }
    }

    hipMalloc((void**)&d_A, N * N * sizeof(int));
    hipMalloc((void**)&d_B, N * N * sizeof(int));
    hipMalloc((void**)&d_C, N * N * sizeof(int));

    hipMemcpy(d_A, A, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * N * sizeof(int), hipMemcpyHostToDevice);

    // Row Addition
    matrixAddRow<<<1, N>>>(d_A, d_B, d_C, N);

    // Column Addition
    // matrixAddCol<<<1, N>>>(d_A, d_B, d_C, N);

    // Element Addition
    // dim3 threadsPerBlock(N, N);
    // matrixAddElement<<<1, threadsPerBlock>>>(d_A, d_B, d_C, N);



    hipMemcpy(C, d_C, N * N * sizeof(int), hipMemcpyDeviceToHost);

    printf("Result: \n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", C[i][j]);
        }
        printf("\n");
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}

// Enter matrix 1: 
// 1 2 3 4
// 1 2 3 4
// 1 2 3 4
// 1 2 3 4

// Enter matrix 2: 
// 5 6 7 8
// 5 6 7 8
// 5 6 7 8
// 5 6 7 8

// Result: 
// 6 8 10 12 
// 6 8 10 12 
// 6 8 10 12 
// 6 8 10 12